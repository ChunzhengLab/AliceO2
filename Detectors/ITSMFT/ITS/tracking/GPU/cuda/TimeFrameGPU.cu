// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
#include <hip/hip_runtime.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

#include "ITStracking/Constants.h"

#include "ITStrackingGPU/Utils.h"
#include "ITStrackingGPU/TimeFrameGPU.h"
#include "ITStrackingGPU/TracerGPU.h"

#include <unistd.h>
#include <thread>
#include <fmt/format.h>

#include "GPUCommonDef.h"
#include "GPUCommonMath.h"
#include "GPUCommonLogger.h"

#ifdef ITS_MEASURE_GPU_TIME
#define START_GPU_STREAM_TIMER(stream, name)           \
  hipEvent_t event_start, event_stop;                 \
  checkGPUError(hipEventCreate(&event_start));        \
  checkGPUError(hipEventCreate(&event_stop));         \
  checkGPUError(hipEventRecord(event_start, stream)); \
  const std::string task_name = name;

#define STOP_GPU_STREAM_TIMER(stream)                                                \
  checkGPUError(hipEventRecord(event_stop, stream));                                \
  checkGPUError(hipEventSynchronize(event_stop));                                   \
  float ms;                                                                          \
  checkGPUError(hipEventElapsedTime(&ms, event_start, event_stop));                 \
  std::cout << "Elapsed time for " << task_name << ": " << ms << " ms" << std::endl; \
  checkGPUError(hipEventDestroy(event_start));                                      \
  checkGPUError(hipEventDestroy(event_stop));
#else
#define START_GPU_STREAM_TIMER(stream, name)
#define STOP_GPU_STREAM_TIMER(stream)
#endif

namespace o2
{
namespace its
{
using constants::GB;
using constants::MB;

namespace gpu
{
using utils::checkGPUError;

void* DefaultGPUAllocator::allocate(size_t size)
{
  LOGP(fatal, "Called DefaultGPUAllocator::allocate with size {}", size);
  return nullptr; // to be implemented
}

template <int nLayers>
TimeFrameGPU<nLayers>::TimeFrameGPU()
{
  mIsGPU = true;
  utils::getDeviceProp(0, true);
}

template <int nLayers>
TimeFrameGPU<nLayers>::~TimeFrameGPU() = default;

template <int nLayers>
void TimeFrameGPU<nLayers>::allocMemAsync(void** ptr, size_t size, Stream* strPtr, bool extAllocator)
{
  if (extAllocator) {
    *ptr = mAllocator->allocate(size);
  } else {
    LOGP(debug, "Calling default CUDA allocator");
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(ptr), size, strPtr->get()));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::setDevicePropagator(const o2::base::PropagatorImpl<float>* propagator)
{
  mPropagatorDevice = propagator;
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadIndexTableUtils(const int iteration)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading indextable utils");
  if (!iteration) {
    LOGP(debug, "gpu-allocation: allocating IndexTableUtils buffer, for {} MB.", sizeof(IndexTableUtils) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mIndexTableUtilsDevice), sizeof(IndexTableUtils), nullptr, getExtAllocator());
  }
  LOGP(debug, "gpu-transfer: loading IndexTableUtils object, for {} MB.", sizeof(IndexTableUtils) / MB);
  checkGPUError(hipMemcpyAsync(mIndexTableUtilsDevice, &mIndexTableUtils, sizeof(IndexTableUtils), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadUnsortedClustersDevice(const int iteration)
{
  if (!iteration) {
    START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading unsorted clusters");
    for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
      LOGP(debug, "gpu-transfer: loading {} unsorted clusters on layer {}, for {} MB.", mUnsortedClusters[iLayer].size(), iLayer, mUnsortedClusters[iLayer].size() * sizeof(Cluster) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mUnsortedClustersDevice[iLayer]), mUnsortedClusters[iLayer].size() * sizeof(Cluster), nullptr, getExtAllocator());
      checkGPUError(hipHostRegister(mUnsortedClusters[iLayer].data(), mUnsortedClusters[iLayer].size() * sizeof(Cluster), hipHostRegisterPortable));
      checkGPUError(hipMemcpyAsync(mUnsortedClustersDevice[iLayer], mUnsortedClusters[iLayer].data(), mUnsortedClusters[iLayer].size() * sizeof(Cluster), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    }
    allocMemAsync(reinterpret_cast<void**>(&mUnsortedClustersDeviceArray), nLayers * sizeof(Cluster*), nullptr, getExtAllocator());
    checkGPUError(hipHostRegister(mUnsortedClustersDevice.data(), nLayers * sizeof(Cluster*), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mUnsortedClustersDeviceArray, mUnsortedClustersDevice.data(), nLayers * sizeof(Cluster*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadClustersDevice(const int iteration)
{
  if (!iteration) {
    START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading sorted clusters");
    for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
      LOGP(debug, "gpu-transfer: loading {} clusters on layer {}, for {} MB.", mClusters[iLayer].size(), iLayer, mClusters[iLayer].size() * sizeof(Cluster) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mClustersDevice[iLayer]), mClusters[iLayer].size() * sizeof(Cluster), nullptr, getExtAllocator());
      checkGPUError(hipHostRegister(mClusters[iLayer].data(), mClusters[iLayer].size() * sizeof(Cluster), hipHostRegisterPortable));
      checkGPUError(hipMemcpyAsync(mClustersDevice[iLayer], mClusters[iLayer].data(), mClusters[iLayer].size() * sizeof(Cluster), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    }
    allocMemAsync(reinterpret_cast<void**>(&mClustersDeviceArray), nLayers * sizeof(Cluster*), nullptr, getExtAllocator());
    checkGPUError(hipHostRegister(mClustersDevice.data(), nLayers * sizeof(Cluster*), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mClustersDeviceArray, mClustersDevice.data(), nLayers * sizeof(Cluster*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadClustersIndexTables(const int iteration)
{
  if (!iteration) {
    START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading sorted clusters");
    for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
      LOGP(debug, "gpu-transfer: loading clusters indextable for layer {} with {} elements, for {} MB.", iLayer, mIndexTables[iLayer].size(), mIndexTables[iLayer].size() * sizeof(int) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mClustersIndexTablesDevice[iLayer]), mIndexTables[iLayer].size() * sizeof(int), nullptr, getExtAllocator());
      checkGPUError(hipMemcpyAsync(mClustersIndexTablesDevice[iLayer], mIndexTables[iLayer].data(), mIndexTables[iLayer].size() * sizeof(int), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    }
    allocMemAsync(reinterpret_cast<void**>(&mClustersIndexTablesDeviceArray), nLayers * sizeof(int), nullptr, getExtAllocator());
    checkGPUError(hipMemcpyAsync(mClustersIndexTablesDeviceArray, mClustersIndexTablesDevice.data(), nLayers * sizeof(int*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createUsedClustersDevice(const int iteration)
{
  if (!iteration) {
    START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "creating used clusters flags");
    for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
      LOGP(debug, "gpu-transfer: creating {} used clusters flags on layer {}, for {} MB.", mUsedClusters[iLayer].size(), iLayer, mUsedClusters[iLayer].size() * sizeof(unsigned char) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mUsedClustersDevice[iLayer]), mUsedClusters[iLayer].size() * sizeof(unsigned char), nullptr, getExtAllocator());
      checkGPUError(hipMemsetAsync(mUsedClustersDevice[iLayer], 0, mUsedClusters[iLayer].size() * sizeof(unsigned char), mGpuStreams[0].get()));
    }
    allocMemAsync(reinterpret_cast<void**>(&mUsedClustersDeviceArray), nLayers * sizeof(unsigned char*), nullptr, getExtAllocator());
    checkGPUError(hipMemcpyAsync(mUsedClustersDeviceArray, mUsedClustersDevice.data(), nLayers * sizeof(unsigned char*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadUsedClustersDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading used clusters flags");
  for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} used clusters flags on layer {}, for {} MB.", mUsedClusters[iLayer].size(), iLayer, mClusters[iLayer].size() * sizeof(unsigned char) / MB);
    checkGPUError(hipMemcpyAsync(mUsedClustersDevice[iLayer], mUsedClusters[iLayer].data(), mUsedClusters[iLayer].size() * sizeof(unsigned char), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadROframeClustersDevice(const int iteration)
{
  if (!iteration) {
    START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading ROframe clusters");
    for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
      LOGP(debug, "gpu-transfer: loading {} ROframe clusters info on layer {}, for {} MB.", mROFramesClusters[iLayer].size(), iLayer, mROFramesClusters[iLayer].size() * sizeof(int) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mROFramesClustersDevice[iLayer]), mROFramesClusters[iLayer].size() * sizeof(int), nullptr, getExtAllocator());
      checkGPUError(hipMemcpyAsync(mROFramesClustersDevice[iLayer], mROFramesClusters[iLayer].data(), mROFramesClusters[iLayer].size() * sizeof(int), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    }
    allocMemAsync(reinterpret_cast<void**>(&mROFrameClustersDeviceArray), nLayers * sizeof(int*), nullptr, getExtAllocator());
    checkGPUError(hipMemcpyAsync(mROFrameClustersDeviceArray, mROFramesClustersDevice.data(), nLayers * sizeof(int*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadTrackingFrameInfoDevice(const int iteration)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading trackingframeinfo");
  if (!iteration) {
    for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
      LOGP(debug, "gpu-transfer: loading {} tfinfo on layer {}, for {} MB.", mTrackingFrameInfo[iLayer].size(), iLayer, mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mTrackingFrameInfoDevice[iLayer]), mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo), nullptr, getExtAllocator());
      checkGPUError(hipHostRegister(mTrackingFrameInfo[iLayer].data(), mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo), hipHostRegisterPortable));
      checkGPUError(hipMemcpyAsync(mTrackingFrameInfoDevice[iLayer], mTrackingFrameInfo[iLayer].data(), mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    }
    allocMemAsync(reinterpret_cast<void**>(&mTrackingFrameInfoDeviceArray), nLayers * sizeof(TrackingFrameInfo*), nullptr, getExtAllocator());
    checkGPUError(hipHostRegister(mTrackingFrameInfoDevice.data(), nLayers * sizeof(TrackingFrameInfo*), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mTrackingFrameInfoDeviceArray, mTrackingFrameInfoDevice.data(), nLayers * sizeof(TrackingFrameInfo*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadMultiplicityCutMask(const int iteration)
{
  if (!iteration) {
    START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading multiplicity cut mask");
    LOGP(debug, "gpu-transfer: loading multiplicity cut mask with {} elements, for {} MB.", mMultiplicityCutMask.size(), mMultiplicityCutMask.size() * sizeof(bool) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mMultMaskDevice), mMultiplicityCutMask.size() * sizeof(uint8_t), nullptr, getExtAllocator());
    checkGPUError(hipMemcpyAsync(mMultMaskDevice, mMultiplicityCutMask.data(), mMultiplicityCutMask.size() * sizeof(uint8_t), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadVertices(const int iteration)
{
  if (!iteration) {
    START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading seeding vertices");
    LOGP(debug, "gpu-transfer: loading {} ROframes vertices, for {} MB.", mROFramesPV.size(), mROFramesPV.size() * sizeof(int) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mROFramesPVDevice), mROFramesPV.size() * sizeof(int), nullptr, getExtAllocator());
    checkGPUError(hipMemcpyAsync(mROFramesPVDevice, mROFramesPV.data(), mROFramesPV.size() * sizeof(int), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    LOGP(debug, "gpu-transfer: loading {} seeding vertices, for {} MB.", mPrimaryVertices.size(), mPrimaryVertices.size() * sizeof(Vertex) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mPrimaryVerticesDevice), mPrimaryVertices.size() * sizeof(Vertex), nullptr, getExtAllocator());
    checkGPUError(hipMemcpyAsync(mPrimaryVerticesDevice, mPrimaryVertices.data(), mPrimaryVertices.size() * sizeof(Vertex), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createTrackletsLUTDevice(const int iteration)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "creating tracklets LUTs");
  for (auto iLayer{0}; iLayer < nLayers - 1; ++iLayer) {
    if (!iteration) {
      LOGP(debug, "gpu-transfer: creating tracklets LUT for {} elements on layer {}, for {} MB.", mClusters[iLayer].size() + 1, iLayer, (mClusters[iLayer].size() + 1) * sizeof(int) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mTrackletsLUTDevice[iLayer]), (mClusters[iLayer].size() + 1) * sizeof(int), nullptr, getExtAllocator());
    }
    checkGPUError(hipMemsetAsync(mTrackletsLUTDevice[iLayer], 0, (mClusters[iLayer].size() + 1) * sizeof(int), mGpuStreams[0].get()));
  }
  if (!iteration) {
    allocMemAsync(reinterpret_cast<void**>(&mTrackletsLUTDeviceArray), (nLayers - 1) * sizeof(int*), nullptr, getExtAllocator());
    checkGPUError(hipMemcpyAsync(mTrackletsLUTDeviceArray, mTrackletsLUTDevice.data(), mTrackletsLUTDevice.size() * sizeof(int*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createTrackletsBuffers()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "creating cells buffers");
  for (auto iLayer{0}; iLayer < nLayers - 1; ++iLayer) {
    mNTracklets[iLayer] = 0;
    checkGPUError(hipMemcpyAsync(&mNTracklets[iLayer], mTrackletsLUTDevice[iLayer] + mClusters[iLayer].size(), sizeof(int), hipMemcpyDeviceToHost));
    LOGP(debug, "gpu-transfer: creating tracklets buffer for {} elements on layer {}, for {} MB.", mNTracklets[iLayer], iLayer, mNTracklets[iLayer] * sizeof(Tracklet) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mTrackletsDevice[iLayer]), mNTracklets[iLayer] * sizeof(Tracklet), nullptr, getExtAllocator());
  }
  allocMemAsync(reinterpret_cast<void**>(&mTrackletsDeviceArray), (nLayers - 1) * sizeof(Tracklet*), nullptr, getExtAllocator());
  checkGPUError(hipHostRegister(mTrackletsDevice.data(), (nLayers - 1) * sizeof(Tracklet*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mTrackletsDeviceArray, mTrackletsDevice.data(), (nLayers - 1) * sizeof(Tracklet*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadTrackletsDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading tracklets");
  for (auto iLayer{0}; iLayer < nLayers - 1; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} tracklets on layer {}, for {} MB.", mTracklets[iLayer].size(), iLayer, mTracklets[iLayer].size() * sizeof(Tracklet) / MB);
    checkGPUError(hipHostRegister(mTracklets[iLayer].data(), mTracklets[iLayer].size() * sizeof(Tracklet), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mTrackletsDevice[iLayer], mTracklets[iLayer].data(), mTracklets[iLayer].size() * sizeof(Tracklet), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadTrackletsLUTDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading tracklets");
  for (auto iLayer{0}; iLayer < nLayers - 2; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading tracklets LUT for {} elements on layer {}, for {} MB", mTrackletsLookupTable[iLayer].size(), iLayer + 1, mTrackletsLookupTable[iLayer].size() * sizeof(int) / MB);
    checkGPUError(hipHostRegister(mTrackletsLookupTable[iLayer].data(), mTrackletsLookupTable[iLayer].size() * sizeof(int), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mTrackletsLUTDevice[iLayer + 1], mTrackletsLookupTable[iLayer].data(), mTrackletsLookupTable[iLayer].size() * sizeof(int), hipMemcpyHostToDevice));
  }
  checkGPUError(hipHostRegister(mTrackletsLUTDevice.data(), (nLayers - 1) * sizeof(int*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mTrackletsLUTDeviceArray, mTrackletsLUTDevice.data(), (nLayers - 1) * sizeof(int*), hipMemcpyHostToDevice));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createNeighboursIndexTablesDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "creating cells neighbours");
  // Here we do also the creation of the CellsDeviceArray, as the cells buffers are populated separately in the previous steps.
  allocMemAsync(reinterpret_cast<void**>(&mCellsDeviceArray), (nLayers - 2) * sizeof(CellSeed*), nullptr, getExtAllocator());
  checkGPUError(hipHostRegister(mCellsDevice.data(), (nLayers - 2) * sizeof(CellSeed*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mCellsDeviceArray, mCellsDevice.data(), (nLayers - 2) * sizeof(CellSeed*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  for (auto iLayer{0}; iLayer < nLayers - 2; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading neighbours LUT for {} elements on layer {}, for {} MB.", mNCells[iLayer], iLayer, mNCells[iLayer] * sizeof(CellSeed) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mNeighboursIndexTablesDevice[iLayer]), (mNCells[iLayer] + 1) * sizeof(int), nullptr, getExtAllocator());
    checkGPUError(hipMemsetAsync(mNeighboursIndexTablesDevice[iLayer], 0, (mNCells[iLayer] + 1) * sizeof(int), mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createNeighboursLUTDevice(const int layer, const unsigned int nCells)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "reserving neighboursLUT");
  LOGP(debug, "gpu-allocation: reserving neighbours LUT for {} elements on layer {} , for {} MB.", nCells + 1, layer, (nCells + 1) * sizeof(int) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mNeighboursLUTDevice[layer]), (nCells + 1) * sizeof(int), nullptr, getExtAllocator()); // We need one element more to move exc -> inc
  checkGPUError(hipMemsetAsync(mNeighboursLUTDevice[layer], 0, (nCells + 1) * sizeof(int), mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadCellsDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading cell seeds");
  for (auto iLayer{0}; iLayer < nLayers - 2; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} cell seeds on layer {}, for {} MB.", mCells[iLayer].size(), iLayer, mCells[iLayer].size() * sizeof(CellSeed) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mCellsDevice[iLayer]), mCells[iLayer].size() * sizeof(CellSeed), nullptr, getExtAllocator());
    allocMemAsync(reinterpret_cast<void**>(&mNeighboursIndexTablesDevice[iLayer]), (mCells[iLayer].size() + 1) * sizeof(int), nullptr, getExtAllocator()); // accessory for the neigh. finding.
    checkGPUError(hipMemsetAsync(mNeighboursIndexTablesDevice[iLayer], 0, (mCells[iLayer].size() + 1) * sizeof(int), mGpuStreams[0].get()));
    checkGPUError(hipMemcpyAsync(mCellsDevice[iLayer], mCells[iLayer].data(), mCells[iLayer].size() * sizeof(CellSeed), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  allocMemAsync(reinterpret_cast<void**>(&mCellsDeviceArray), (nLayers - 2) * sizeof(CellSeed*), nullptr, getExtAllocator());
  checkGPUError(hipMemcpyAsync(mCellsDeviceArray, mCellsDevice.data(), (nLayers - 2) * sizeof(CellSeed*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createCellsLUTDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "creating cells LUTs");
  for (auto iLayer{0}; iLayer < nLayers - 2; ++iLayer) {
    LOGP(debug, "gpu-transfer: creating cell LUT for {} elements on layer {}, for {} MB.", mNTracklets[iLayer] + 1, iLayer, (mNTracklets[iLayer] + 1) * sizeof(int) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mCellsLUTDevice[iLayer]), (mNTracklets[iLayer] + 1) * sizeof(int), nullptr, getExtAllocator());
    checkGPUError(hipMemsetAsync(mCellsLUTDevice[iLayer], 0, (mNTracklets[iLayer] + 1) * sizeof(int), mGpuStreams[0].get()));
  }
  allocMemAsync(reinterpret_cast<void**>(&mCellsLUTDeviceArray), (nLayers - 2) * sizeof(int*), nullptr, getExtAllocator());
  checkGPUError(hipMemcpyAsync(mCellsLUTDeviceArray, mCellsLUTDevice.data(), mCellsLUTDevice.size() * sizeof(int*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createCellsBuffers(const int layer)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "creating cells buffers");
  mNCells[layer] = 0;
  checkGPUError(hipMemcpyAsync(&mNCells[layer], mCellsLUTDevice[layer] + mNTracklets[layer], sizeof(int), hipMemcpyDeviceToHost));
  LOGP(debug, "gpu-transfer: creating cell buffer for {} elements on layer {}, for {} MB.", mNCells[layer], layer, mNCells[layer] * sizeof(CellSeed) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mCellsDevice[layer]), mNCells[layer] * sizeof(CellSeed), nullptr, getExtAllocator());

  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadCellsLUTDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading cells LUTs");
  for (auto iLayer{0}; iLayer < nLayers - 3; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading cell LUT for {} elements on layer {}, for {} MB.", mCellsLookupTable[iLayer].size(), iLayer, mCellsLookupTable[iLayer].size() * sizeof(int) / MB);
    checkGPUError(hipHostRegister(mCellsLookupTable[iLayer].data(), mCellsLookupTable[iLayer].size() * sizeof(int), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mCellsLUTDevice[iLayer + 1], mCellsLookupTable[iLayer].data(), mCellsLookupTable[iLayer].size() * sizeof(int), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadRoadsDevice()
{
  LOGP(debug, "gpu-transfer: loading {} roads, for {} MB.", mRoads.size(), mRoads.size() * sizeof(Road<nLayers - 2>) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mRoadsDevice), mRoads.size() * sizeof(Road<nLayers - 2>), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipHostRegister(mRoads.data(), mRoads.size() * sizeof(Road<nLayers - 2>), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mRoadsDevice, mRoads.data(), mRoads.size() * sizeof(Road<nLayers - 2>), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadTrackSeedsDevice(std::vector<CellSeed>& seeds)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "loading track seeds");
  LOGP(debug, "gpu-transfer: loading {} track seeds, for {} MB.", seeds.size(), seeds.size() * sizeof(CellSeed) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mTrackSeedsDevice), seeds.size() * sizeof(CellSeed), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipHostRegister(seeds.data(), seeds.size() * sizeof(CellSeed), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mTrackSeedsDevice, seeds.data(), seeds.size() * sizeof(CellSeed), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createNeighboursDevice(const unsigned int& layer, std::vector<std::pair<int, int>>& neighbours)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "reserving neighbours");
  mCellsNeighbours[layer].clear();
  mCellsNeighbours[layer].resize(neighbours.size());
  LOGP(debug, "gpu-allocation: reserving {} neighbours (pairs), for {} MB.", neighbours.size(), neighbours.size() * sizeof(gpuPair<int, int>) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mNeighbourPairsDevice[layer]), neighbours.size() * sizeof(gpuPair<int, int>), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipMemsetAsync(mNeighbourPairsDevice[layer], -1, neighbours.size() * sizeof(gpuPair<int, int>), mGpuStreams[0].get()));
  LOGP(debug, "gpu-allocation: reserving {} neighbours, for {} MB.", neighbours.size(), neighbours.size() * sizeof(gpuPair<int, int>) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mNeighboursDevice[layer]), neighbours.size() * sizeof(int), &(mGpuStreams[0]), getExtAllocator());
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createNeighboursDeviceArray()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "reserving neighbours");
  allocMemAsync(reinterpret_cast<void**>(&mNeighboursDeviceArray), (nLayers - 2) * sizeof(int*), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipMemcpyAsync(mNeighboursDeviceArray, mNeighboursDevice.data(), (nLayers - 2) * sizeof(int*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createTrackITSExtDevice(std::vector<CellSeed>& seeds)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "reserving tracks");
  mTrackITSExt.clear();
  mTrackITSExt.resize(seeds.size());
  LOGP(debug, "gpu-allocation: reserving {} tracks, for {} MB.", seeds.size(), seeds.size() * sizeof(o2::its::TrackITSExt) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mTrackITSExtDevice), seeds.size() * sizeof(o2::its::TrackITSExt), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipMemsetAsync(mTrackITSExtDevice, 0, seeds.size() * sizeof(o2::its::TrackITSExt), mGpuStreams[0].get()));
  checkGPUError(hipHostRegister(mTrackITSExt.data(), seeds.size() * sizeof(o2::its::TrackITSExt), hipHostRegisterPortable));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadCellsDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "downloading cells");
  for (int iLayer{0}; iLayer < nLayers - 2; ++iLayer) {
    LOGP(debug, "gpu-transfer: downloading {} cells on layer: {}, for {} MB.", mNCells[iLayer], iLayer, mNCells[iLayer] * sizeof(CellSeed) / MB);
    mCells[iLayer].resize(mNCells[iLayer]);
    checkGPUError(hipMemcpyAsync(mCells[iLayer].data(), mCellsDevice[iLayer], mNCells[iLayer] * sizeof(CellSeed), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadCellsLUTDevice()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "downloading cell luts");
  for (auto iLayer{0}; iLayer < nLayers - 3; ++iLayer) {
    LOGP(debug, "gpu-transfer: downloading cells lut on layer {} for {} elements", iLayer, (mNTracklets[iLayer + 1] + 1));
    mCellsLookupTable[iLayer].resize(mNTracklets[iLayer + 1] + 1);
    checkGPUError(hipMemcpyAsync(mCellsLookupTable[iLayer].data(), mCellsLUTDevice[iLayer + 1], (mNTracklets[iLayer + 1] + 1) * sizeof(int), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
  }
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadCellsNeighboursDevice(std::vector<std::vector<std::pair<int, int>>>& neighbours, const int layer)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), fmt::format("downloading neighbours from layer {}", layer));
  LOGP(debug, "gpu-transfer: downloading {} neighbours, for {} MB.", neighbours[layer].size(), neighbours[layer].size() * sizeof(std::pair<int, int>) / MB);
  // TODO: something less dangerous than assuming the same memory layout of std::pair and gpuPair... or not? :)
  checkGPUError(hipMemcpyAsync(neighbours[layer].data(), mNeighbourPairsDevice[layer], neighbours[layer].size() * sizeof(gpuPair<int, int>), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadNeighboursLUTDevice(std::vector<int>& lut, const int layer)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), fmt::format("downloading neighbours LUT from layer {}", layer));
  LOGP(debug, "gpu-transfer: downloading neighbours LUT for {} elements on layer {}, for {} MB.", lut.size(), layer, lut.size() * sizeof(int) / MB);
  checkGPUError(hipMemcpyAsync(lut.data(), mNeighboursLUTDevice[layer], lut.size() * sizeof(int), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadTrackITSExtDevice(std::vector<CellSeed>& seeds)
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "downloading tracks");
  LOGP(debug, "gpu-transfer: downloading {} tracks, for {} MB.", mTrackITSExt.size(), mTrackITSExt.size() * sizeof(o2::its::TrackITSExt) / MB);
  checkGPUError(hipMemcpyAsync(mTrackITSExt.data(), mTrackITSExtDevice, seeds.size() * sizeof(o2::its::TrackITSExt), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
  checkGPUError(hipHostUnregister(mTrackITSExt.data()));
  checkGPUError(hipHostUnregister(seeds.data()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::unregisterRest()
{
  START_GPU_STREAM_TIMER(mGpuStreams[0].get(), "unregistering rest of the host memory");
  LOGP(debug, "unregistering rest of the host memory...");
  checkGPUError(hipHostUnregister(mCellsDevice.data()));
  checkGPUError(hipHostUnregister(mTrackletsDevice.data()));
  STOP_GPU_STREAM_TIMER(mGpuStreams[0].get());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::unregisterHostMemory(const int maxLayers)
{
  for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
    checkGPUError(hipHostUnregister(mUnsortedClusters[iLayer].data()));
    checkGPUError(hipHostUnregister(mClusters[iLayer].data()));
    checkGPUError(hipHostUnregister(mTrackingFrameInfo[iLayer].data()));
  }
  checkGPUError(hipHostUnregister(mTrackingFrameInfoDevice.data()));
  checkGPUError(hipHostUnregister(mUnsortedClustersDevice.data()));
  checkGPUError(hipHostUnregister(mClustersDevice.data()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initialise(const int iteration,
                                       const TrackingParameters& trkParam,
                                       const int maxLayers,
                                       IndexTableUtils* utils,
                                       const TimeFrameGPUParameters* gpuParam)
{
  mGpuStreams.resize(mGpuParams.nTimeFrameChunks);
  o2::its::TimeFrame::initialise(iteration, trkParam, maxLayers);
}

template class TimeFrameGPU<7>;
} // namespace gpu
} // namespace its
} // namespace o2
